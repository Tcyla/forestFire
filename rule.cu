#include "hip/hip_runtime.h"
char get_torified_coeffs( int i ,  int j, char** matrix, int row, int col )
{
	// The grid is a torus. This function returns 
	//
	int x,y;

	if ( i == -1 )
		x = row - 1;
	else if ( i == row )
		x = 0;
	else
		x = i;

	if ( j == -1 )
		y = col - 1;
	else if ( j == col )
		y = 0;
	else
		y = j;

	return (matrix)[x][y];
}

char rule(int i, int j, char** matrix, int row, int col )
{
	/*
	 * rules are:
	 * 1) if a fire tile is in the neighbourhood of a forest tile then
	 *    the forest tile is set on fire.
	 * 2) if fire then ash.
	 */

	char res;
	try
	{
		res = (matrix)[i][j];
	}
	catch (const char* e)
	{
		throw e;
	}
	// table of the surrounding cells
	char tab[8] = {
		get_torified_coeffs(i-1, j-1, matrix, row, col), get_torified_coeffs(i-1, j, matrix, row, col),
		get_torified_coeffs(i-1, j+1, matrix, row, col), get_torified_coeffs(i, j-1, matrix, row, col),
		get_torified_coeffs(i, j+1, matrix, row, col), get_torified_coeffs(i+1, j-1, matrix, row, col),
		get_torified_coeffs(i+1, j, matrix, row, col), get_torified_coeffs(i+1, j+1, matrix, row, col)
	};

	// counting the living cells
	bool fire_around = false;
	for (int k = 0; k < 8; ++k)
	{
		if ( tab[k] == 'f' )
		{
			fire_around = true;
			break;
		}
	}

	// checking if the cell lives
	if ( (matrix)[i][j] == 'w' && fire_around )
	{
		res = 'f';
	}else if ( (matrix)[i][j] == 'f' )
	{
		res = 'a';
	}

	return res;
}

__global__
void rule_application(char** matrix[], int row, int col )
{
    int nThreadRef;
    // int nThreadRef = cudathreadnumber
    int i = nThreadRef / col;
    int j = nThreadRef % col;
    (*matrix)[i][j] = rule( i, j, *matrix, row, col );
}

void rule_applied( char** matrix[], int row, int col )
{
    int nbThread = row * col;
    rule_application<<<nbThread>>>(char** matrix[], int row, int col); 
}